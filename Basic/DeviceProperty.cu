#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                \
{  const hipError_t error = call;                 \
   if (error != hipSuccess)                       \
  {                                                \
    printf("Error: %s:%d, ", __FILE__, __LINE__);  \
    printf("code:%d, reason: %s\n", error,         \
                     hipGetErrorString(error));   \
    exit(1);                                       \
  }                                                \
}

int getDeviceProperty() {

  int device;

  const float KILO = 1000.f;
  const float MEGA = 1000.f*1000.f;
  const float GIGA = 1000.f*1000.f*1000.f;

  CHECK(hipGetDeviceCount(&device));

  for (int i = 0; i < device; i++) {
    /* Get device properties function call */
    hipDeviceProp_t property;
    CHECK(hipGetDeviceProperties(&property, i));

    printf("<~~~~~~~~~ Device ~~~~~~~~~>\n");
    printf("ID: %d\n", device);
    printf("Name %s\n", property.name);
    printf("Compute Capability %d.%d\n", property.major, property.minor);
    printf("Clock Rate %.2f Mhz\n", (property.clockRate)/(KILO));
    printf("Memory Clock Rate %.2f Ghz\n", (property.memoryClockRate)/(MEGA));
    printf("Total Global Memory %.2f GB\n", property.totalGlobalMem/(GIGA));
    printf("Total Const Memory %.2f KB\n", property.totalConstMem/(KILO));
    printf("Shared Memory Per Block %.2f KB\n", property.sharedMemPerBlock/(KILO));
  }

  return 0;
}
